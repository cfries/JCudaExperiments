
#include <hip/hip_runtime.h>
extern "C"
__global__ void cuAdd(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] + b[i];
    }

}

extern "C"
__global__ void cuMult(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] * b[i];
    }

}

extern "C"
__global__ void cuDiv(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = a[i] / b[i];
    }

}

extern "C"
__global__ void cuExp(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        result[i] = expf(a[i]);
    }

}

